#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <random>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

using namespace std;
using namespace nvcuda;
// from https://github.com/jarro2783/cxxopts
#include "cxxopts.hpp"

#define cudaCheck(err) (cudaErrorCheck(err, __FILE__, __LINE__))
#define cublasCheck(err) (cublasErrorCheck(err, __FILE__, __LINE__))
#define ROUND_UP_TO_NEAREST(M, N) (((M) + (N)-1) / (N))

//using namespace nvcuda;

enum Algo
{
    cublas_hgemm = 0,
    cuda_hgemm,
    tensor_hgemm,
    numAlgos
};

const char *algo2str(Algo a)
{
    switch (a)
    {
    case cublas_hgemm:
        return "cublas_hgemm";
    /*case cuda_hgemm:
        return "cuda_hgemm";*/
    case tensor_hgemm:
        return "tensor_hgemm";
    default:
        return "INVALID";
    }
}

void cudaErrorCheck(hipError_t error, const char *file, int line);
void cublasErrorCheck(hipblasStatus_t status, const char *file, int line);
void randomize_matrix(half *mat, int N);
void const_init_matrix(half *mat, int N, half F);
bool verify_matrix(half *expected, half *actual, int M, int N);
void print_matrix(const half *A, int M, int N, std::ostream &outs);
void runAlgo(Algo algo, hipblasHandle_t handle, int M, int N, int K, half alpha, half *A, half *B, half beta, half *C);
void runCublas(hipblasHandle_t handle, int M, int N, int K, half alpha, half *A, half *B, half beta, half *C);

const std::string errLogFile = "gemmValidationFailure.txt";

// NB: must use a single generator to avoid duplicates
std::default_random_engine generator(2);
std::uniform_real_distribution<float> distribution(0, 1);

int main(int argc, char **argv)
{
    // command-line flags
    cxxopts::Options options("gemm.cu", "CUDA GEMM kernels");
    options.add_options()("size", "matrix size (N x N)", cxxopts::value<uint16_t>()->default_value("128"))                //
        ("reps", "repeat GEMM this many times", cxxopts::value<uint16_t>()->default_value("1"))                           //
        ("algo", "GEMM algorithm to use, a number in [0,2], 0 is cuBLAS", cxxopts::value<uint16_t>()->default_value("0")) //
        ("validate", "Validate output against cuBLAS", cxxopts::value<bool>()->default_value("true"))                     //
        ("rngseed", "PRNG seed", cxxopts::value<uint>()->default_value("2"))                     //
        ("h,help", "Print usage");

    auto clFlags = options.parse(argc, argv);
    if (clFlags.count("help"))
    {
        std::cout << options.help() << std::endl;
        exit(0);
    }
    const uint16_t SIZE = clFlags["size"].as<uint16_t>();
    if (SIZE % 32 != 0)
    {
        //std::cout << "--size must be a multiple of 32" << std::endl;
        //exit(EXIT_FAILURE);
    }
    const uint16_t REPS = clFlags["reps"].as<uint16_t>();
    const Algo ALGO = static_cast<Algo>(clFlags["algo"].as<uint16_t>());
    if (ALGO >= numAlgos)
    {
        printf("Invalid algorithm: %d\n", ALGO);
        exit(EXIT_FAILURE);
    }

    const bool VALIDATE = clFlags["validate"].as<bool>();
    const uint SEED = clFlags["rngseed"].as<uint>();
    generator.seed(SEED);
    printf("Multiplying two %u x %u matrices with %u trials using %s algorithm\n", SIZE, SIZE, REPS, algo2str(ALGO));

    cudaCheck(hipSetDevice(0));

    // Setup cublas
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetMathMode(handle,HIPBLAS_TENSOR_OP_MATH);

    // Using cudaEvent for gpu stream timing, cudaEvent is equivalent to
    // publishing event tasks in the target stream
    hipEvent_t beg, end;
    cudaCheck(hipEventCreate(&beg));
    cudaCheck(hipEventCreate(&end));

    uint16_t m = SIZE, n = SIZE, k = SIZE;

    // GEMM computes C = α*AB+β*C

    // just do pure A*B (for simpler debugging)
    half alpha = 1.0, beta = 0.0, initC = 1.0;

    half *A = nullptr, *B = nullptr, *C = nullptr, *C_ref = nullptr;     // host matrices
    half *dA = nullptr, *dB = nullptr, *dC = nullptr, *dC_ref = nullptr; // device matrices

    A = (half *)malloc(sizeof(half) * SIZE * SIZE);
    B = (half *)malloc(sizeof(half) * SIZE * SIZE);
    C = (half *)malloc(sizeof(half) * SIZE * SIZE);
    C_ref = (half *)malloc(sizeof(half) * SIZE * SIZE);

    randomize_matrix(A, SIZE * SIZE);
    randomize_matrix(B, SIZE * SIZE);
    randomize_matrix(C, SIZE * SIZE);

    const_init_matrix(C, SIZE * SIZE, initC);


    cudaCheck(hipMalloc((void **)&dA, sizeof(half) * SIZE * SIZE));
    cudaCheck(hipMalloc((void **)&dB, sizeof(half) * SIZE * SIZE));
    cudaCheck(hipMalloc((void **)&dC, sizeof(half) * SIZE * SIZE));
    cudaCheck(hipMalloc((void **)&dC_ref, sizeof(half) * SIZE * SIZE));

    cudaCheck(hipMemcpy(dA, A, sizeof(half) * SIZE * SIZE, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dB, B, sizeof(half) * SIZE * SIZE, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dC, C, sizeof(half) * SIZE * SIZE, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(dC_ref, C, sizeof(half) * SIZE * SIZE, hipMemcpyHostToDevice));

    printf("dimensions(m=n=k) %u, alpha: %f, beta: %f\n", m, __half2float(alpha), __half2float(beta));

    // Verify the correctness of the calculation, and execute it once before the
    // kernel function timing to avoid cold start errors
    if (!VALIDATE)
    {
        printf("disabled validation\n");
    }
    else
    {
        // run cublas to get correct answer in dC_ref
        runCublas(handle, m, n, k, alpha, dA, dB, beta, dC_ref);

        // run user's algorithm, filling in dC
        runAlgo(ALGO, handle, m, n, k, alpha, dA, dB, beta, dC);

        cudaCheck(hipDeviceSynchronize());

        // copy both results back to host
        hipMemcpy(C, dC, sizeof(half) * m * n, hipMemcpyDeviceToHost);
        hipMemcpy(C_ref, dC_ref, sizeof(half) * m * n, hipMemcpyDeviceToHost);

        if (verify_matrix(C_ref, C, n, m))
        {
            printf("Validated successfully!\n");
        }
        else
        {
            printf("Failed validation against NVIDIA cuBLAS.\n");
            std::cout << " Logging faulty output into " << errLogFile << "\n";
            std::ofstream fs;
            fs.open(errLogFile, std::ios::out | std::ios::trunc);
            fs << "α=" << __half2float(alpha) << " β=" << __half2float(beta) << std::endl;
            fs << "C matrix initialized to " << __half2float(initC) << std::endl << std::endl;
            fs << "A:" << std::endl;
            print_matrix(A, m, n, fs);
            fs << "B:" << std::endl;
            print_matrix(B, m, n, fs);
            fs << "C:" << std::endl;
            print_matrix(C, m, n, fs);
            fs << "Expected:" << std::endl;
            print_matrix(C_ref, m, n, fs);
            fs.close();
            exit(EXIT_FAILURE);
        }
    }

    // timing run(s)
    hipEventRecord(beg);
    for (int j = 0; j < REPS; j++)
    {
        // We don't reset dC between runs to save time
        runAlgo(ALGO, handle, m, n, k, alpha, dA, dB, beta, dC);
        cudaCheck(hipDeviceSynchronize());
    }

    // TODO: measure timing without memory transfers?
    cudaCheck(hipEventRecord(end));
    cudaCheck(hipEventSynchronize(beg));
    cudaCheck(hipEventSynchronize(end));
    float elapsed_time;
    cudaCheck(hipEventElapsedTime(&elapsed_time, beg, end));
    elapsed_time /= 1000.; // Convert to seconds

    double flops = (double)2 * m * n * k;
    printf(
        "Average elapsed time: (%7.6f) s, performance: (%7.2f) GFLOPS. size: (%u).\n",
        elapsed_time / REPS,
        (REPS * flops * 1e-9) / elapsed_time,
        m);

    // free CPU and GPU memory
    free(A);
    free(B);
    free(C);
    free(C_ref);
    cudaCheck(hipFree(dA));
    cudaCheck(hipFree(dB));
    cudaCheck(hipFree(dC));
    cudaCheck(hipFree(dC_ref));
    cublasCheck(hipblasDestroy(handle));

    return 0;
}

/** Function to check for errors in CUDA API calls */
void cudaErrorCheck(hipError_t error, const char *file, int line)
{
    if (error != hipSuccess)
    {
        printf("[CUDA ERROR] at file %s:%d:\n%s: %s\n", file, line,
               hipGetErrorName(error), hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
};

void cublasErrorCheck(hipblasStatus_t status, const char *file, int line)
{
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        printf("[CUDA ERROR] at file %s:%d:\n %s: %s\n", file, line,
               cublasGetStatusName(status), cublasGetStatusString(status));
        exit(EXIT_FAILURE);
    }
}

/** Initialize the given matrix `mat` which has `N` contiguous values. Contents of `mat` are set to random values. */
void randomize_matrix(half *mat, int N)
{
    for (int i = 0; i < N; i++)
    {
        mat[i] = distribution(generator);
    }
}

void const_init_matrix(half *mat, int N, half F)
{
    for (int i = 0; i < N; i++)
    {
        mat[i] = F;
    }
}

/** Print the given MxN matrix `mat` to the provided output stream. */
void print_matrix(const half *A, int M, int N, std::ostream &outs)
{
    outs << "[";
    for (int i = 0; i < M * N; i++)
    {
        if ((i + 1) % N == 0)
        {
            outs << std::fixed << std::setprecision(3) << __half2float(A[i]);
        }
        else
        {
            outs << std::fixed << std::setprecision(3) << __half2float(A[i]) << ", ";
        }
        if ((i + 1) % N == 0)
        {
            if (i + 1 < M * N)
                outs << ";" << std::endl;
        }
    }
    outs << "]" << std::endl << std::endl;
}

bool verify_matrix(half *expected, half *actual, int M, int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < M; j++)
        {
            half fexp = (expected[(i * N) + j]);
            half fact = (actual[(i * N) + j]);
            double diff = std::fabs(__half2float(fexp) - __half2float(fact));
            if (diff > 0.002)
            {
                printf("Divergence! Should be %5.3f, is %5.3f (diff %5.3f) at [%d,%d]\n",
                       __half2float(fexp), __half2float(fact), __half2float(diff), i, j);
                return false;
            }
        }
    }
    return true;
}

void runCublas(hipblasHandle_t handle, int M, int N, int K, half alpha,
               half *A, half *B, half beta, half *C)
{

    hipblasStatus_t ok = hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, M, K, &alpha, B, HIP_R_16F, K, A, HIP_R_16F, K,
                                    &beta, C, HIP_R_16F, N, HIPBLAS_COMPUTE_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    cublasCheck(ok);
}

/*__global__ void runBasic(int M, int N, int K, float alpha, float *A, float *B, float beta, float *C)
{
    const unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < M && y < N)
    {
        float tmp = 0.0;
        // C = α*(AxB)+β*C
        for (int i = 0; i < K; ++i)
        {
            // tmp += __A__[x][i] * __B__[i][y]
            tmp += A[(x * K) + i] * B[(i * N) + y];
        }
        // __C__[x][y]
        C[(x * N) + y] = (alpha * tmp) + (beta * C[x * N + y]);
    }
}
*/

#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

__global__ void tensor_impl(int M, int N, int K, half alpha, half *A, half *B, half beta, half *C)
{
    

    int K_tiles = ROUND_UP_TO_NEAREST(K,WMMA_K);

    int row  = blockIdx.y * WMMA_M;
    int column  = blockIdx.x * WMMA_N;

    if(row >= M && column >= N)
    {
        return;
    }

    nvcuda::wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> C_frag;

    nvcuda::wmma::fill_fragment(C_frag,0.0);

#pragma unroll
    for(int i = 0; i<K_tiles; ++i)
    {
        nvcuda::wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> A_frag;
        nvcuda::wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> B_frag;

        nvcuda::wmma::load_matrix_sync(A_frag, A + row*K + i*WMMA_K,K);
        nvcuda::wmma::load_matrix_sync(B_frag, B + i*WMMA_K + column*K,K);

        nvcuda::wmma::mma_sync(C_frag, A_frag, B_frag, C_frag);

    }

    wmma::store_matrix_sync(C + row * N + column, C_frag, N, wmma::mem_row_major);


}

/*

__global__ void runSharedMem(int M, int N, int K, half alpha, half *A, half *B, half beta, half *C)
{
    // HW2 TODO: Use shared memory to cache square FxF tiles of the A and B matrices in shared memory 
    // (SA and SB, respectively, provided below). Each thread should compute the result for one cell 
    // of the output matrix C.

    // Note, you will also need to change the grid dimensions in the kernel launch below to take into account the value
    // of F (which is a constant, defined above). You should experiment with different values of F to see how it 
    // affects performance.

    __shared__ half SA[F][F];
    __shared__ half SB[F][F];

    const unsigned blkidx = blockIdx.x;
    const unsigned blkidy = blockIdx.y;
    const unsigned bdimx = blockDim.x;
    const unsigned bdimy = blockDim.x;
    const unsigned threadx = threadIdx.x;
    const unsigned thready = threadIdx.y;

    const unsigned column = blockIdx.x * F + threadIdx.x;
    const unsigned row = blockIdx.y * F + threadIdx.y;

    // const unsigned row = blkidy * F + thready;
    // const unsigned column = blkidx * F + threadx;
    if (row < M && column < N)
    {
        half tmp = 0.0;
        // C = α*(AxB)+β*C
        for (int i = 0; i < K/F; ++i)
        {
            // tmp += __A__[x][i] * __B__[i][y]
            //tmp += A[(x * K) + i] * B[(i * N) + y];
            SA[thready][threadx] = A[(row * K) + i*F+threadx];

            //Print debugs
           //printf("A element at address %d = %f \n",(row * K) + i*F+threadx, A[(row * K) + i*F+threadx]);
           //printf(" A Accessed into SA (address: %d,%d): %f , row  = %d , blockid = %d \n",thready,threadx,SA[thready][threadx],row,blockIdx.y);

            //Print debugs
            SB[thready][threadx] = B[(i* F + thready)*K + column];
           //printf("B element at address %d  = %f \n",(i* F + thready)*K + column, B[(i* F + thready)*K + column]);
           //printf(" B Accessed into SA (address: %d,%d): %f \n",thready,threadx,SB[thready][threadx]);

           __syncthreads();

            for(int j = 0; j < F; ++j)
            {
                tmp += SA[thready][j] * SB[j][threadx];

            }
                __syncthreads();
        }
        // __C__[x][y]
        C[(row * K) + column] = (alpha * tmp) + (beta * C[row * K + column]);
    }
    }
*/
/*
const uint G = 4;
const uint F = 32;

__global__ void run_cuda_hgemm(int M, int N, int K, half alpha, half *A, half *B, half beta, half *C)
{
    // HW3 TODO: Copy your runSharedMem() code here and update it so that each thread computes the result for GxG cells 
    // of the output matrix C. Each thread should accumulate temporary results in the local LC matrix, provided below,
    // before writing them to C in global memory.

    // Note, you will also need to change the grid dimensions in the kernel launch below. You should experiment 
    // with different values of F and G to see how they affect performance.

    __shared__ half SA[F][F];
    __shared__ half SB[F][F];

    float LC[G][G] = {0.0};
    half resSA[G] = {0.0}; // Temp 
    half resSB[G] = {0.0}; // Temp

    //const unsigned threadx = threadIdx.x;
    //const unsigned thready = threadIdx.y;

   // const unsigned column = blockIdx.x * F + threadIdx.x;
   // const unsigned row = blockIdx.y * F + threadIdx.y;

    int divider = (F*F)/(G*G);
    int stride  = divider / F;

    const unsigned row_s = threadIdx.x / F;
    const unsigned column_s = threadIdx.x % F;
    const unsigned row_l = (threadIdx.x / (F/G));
    const unsigned column_l = (threadIdx.x % (F/G)) ;

    // Start of the Matrices
    A += blockIdx.y * F * K;
    B += blockIdx.x * F;
    C += blockIdx.y * F * N + blockIdx.x * F;
    


        for (int i = 0; i < K/F; ++i)
        {
            // tmp += __A__[x][i] * __B__[i][y]
            for(int c = 0; c < F; c+=stride)
            {
                SA[row_s + c][column_s] = A[((row_s + c) * K) + i*F+column_s];
                SB[row_s + c][column_s] = B[(i* F + (row_s + c))*K + column_s];
                
            }
            __syncthreads();    

            for(int p = 0 ; p < F ; ++p)
            {
                for(int r = 0; r<G; ++r)
                {
                    resSA[r] = SA[row_l * G + r][p];
                    resSB[r] = SB[p][column_l*G +r];
                    //printf("row = %d, column = %d \n",row_l,column_l);

                }

                for(int m = 0 ; m < G ; ++m)
                {
                    for(int n = 0 ; n < G ; ++n)
                    {
                        LC[m][n] +=  __half2float(resSA[m]) * __half2float(resSB[n]);

                        //printf("\n LC  = %f \n", LC[m][n]);
                    }
                    
                }
                
            }
    
            __syncthreads();
        }
        // __C__[x][y]

        for(int m = 0 ; m < G ; ++m)
        {
            for(int n = 0 ; n < G ; ++n)
            {
                C[(row_l * G + m)*M + column_l * G + n] = (alpha * (LC[m][n])) + (beta * C[(row_l * G + m)*M + column_l * G + n]);
                //printf("\n C at %d,%d  = %f , row  = %d, column = %d \n", (row_l * G + m)*M , column_l * G + n ,C[(row_l * G + m)*M + column_l * G + n], row_l, column_l);
            }
        }
        
}*/


void runAlgo(Algo algo, hipblasHandle_t handle, int M, int N, int K, half alpha,
             half *A, half *B, half beta, half *C)
{
    switch (algo)
    {
    case cublas_hgemm:
        runCublas(handle, M, N, K, alpha, A, B, beta, C);
        break;
    // case cuda_hgemm:
    // {
    //     assert(0 == M % F);
    //     assert(0 == N % F);
    //     assert(0 == K % F);
    //     assert(0 == F % G);
    //     assert((F*F) / (G*G) >= F);
    //     // TODO: update your grid here
    //     dim3 gridDim(ROUND_UP_TO_NEAREST(M, F), ROUND_UP_TO_NEAREST(N, F));
    //     //dim3 gridDim(ROUND_UP_TO_NEAREST(M, 32)/F, ROUND_UP_TO_NEAREST(N, 32)/F);
    //     dim3 blockDim((F*F)/(G*G));
    //     run_cuda_hgemm<<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
    //     break;
    // }
    case tensor_hgemm:
    {   
        dim3 block(32);
        dim3 grid(ROUND_UP_TO_NEAREST(N, WMMA_N), ROUND_UP_TO_NEAREST(M,WMMA_M));

        tensor_impl<<<grid, block>>>(M, N, K, alpha, A, B, beta, C);
        break;
    }
    default:
        printf("Invalid algorithm: %d\n", algo);
        exit(EXIT_FAILURE);
    }
    cudaCheck(hipDeviceSynchronize()); // wait for kernel to finish
    cudaCheck(hipGetLastError());      // check for errors from kernel run
}
